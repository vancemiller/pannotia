#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  lud.cu
 *
 *    Description:  The main wrapper for the suite
 *
 *        Version:  1.0
 *        Created:  10/22/2009 08:40:34 PM
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  Liang Wang (lw2aw), lw2aw@virginia.edu
 *        Company:  CS@UVa
 *
 * =====================================================================================
 */

#include <argp.h>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <errno.h>
#include <unistd.h>

extern "C" {
#include "lud_kernel.cuh"
}
#include "hip/hip_runtime_api.h"

#ifdef RD_WG_SIZE_0_0
#define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
#define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
#define BLOCK_SIZE RD_WG_SIZE
#else
#define BLOCK_SIZE 16
#endif

#include "../timing.h"

#define VPRINT(verbose, format, ...) \
  if (verbose) {\
    fprintf(stdout, format, ## __VA_ARGS__);\
  }

////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                                                                //
//                                       Argument processing                                      //
//                                                                                                //
////////////////////////////////////////////////////////////////////////////////////////////////////

static char doc[] = "Rodinia LUD Benchmark";
static char args_doc[] = "";

static struct argp_option options[] = {
  {"device", 'd', "DEVICE", 0, "CUDA Device ID"},
  {"file", 'f', "FILEPATH", 0, "Path to file containing input data."},
  {"size", 's', "SIZE", 0, "Generate input with SIZE elements. (Ignores file input)"},
  {"unified", 'u', 0, 0, "Use unified memory"},
  {"verbose", 'v', 0, 0, "Verbose output"},
  {0},
};

struct arguments {
  uint8_t device;
  char* file;
  uint32_t size;
  bool unified;
  bool verbose;
};

static error_t parse_opt(int key, char* arg, struct argp_state* state) {
  struct arguments* args = (struct arguments*) state->input;
  switch (key) {
    case 'd':
      args->device = (int) strtol(arg, NULL, 0);
      break;
    case 'f':
      args->file = arg;
      break;
    case 's':
      args->size = (int) strtol(arg, NULL, 0);
      break;
    case 'u':
      args->unified = true;
      break;
    case 'v':
      args->verbose = true;
      break;
    default:
      return ARGP_ERR_UNKNOWN;
  }
  return 0;
}

static struct argp argp = {options, parse_opt, args_doc, doc};

////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                                                                //
//                                      Forward declarations                                      //
//                                                                                                //
////////////////////////////////////////////////////////////////////////////////////////////////////

void print_matrix(float *m, int size);
long long lud_cuda(float *m, int size, hipStream_t stream);

////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                                                                //
//                                              Main                                              //
//                                                                                                //
////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv) {
  struct arguments args;
  // Defaults
  args.device = 0;
  args.file = NULL;
  args.size = 0;
  args.unified = false;
  args.verbose = false;
  // Parse command line arguments
  argp_parse(&argp, argc, argv, 0, 0, &args);
  if (!args.size && !args.file) {
    fprintf(stderr, "Provide -s or -f flag. Use --help for help\n");
    exit(EXIT_FAILURE);
  }

  VPRINT(args.verbose, "WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

  size_t size = args.size;

  float* m;
  float* d_m;

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  float time_pre = 0;
  float time_post = 0;
  float time_serial = 0;
  float time_copy_in = 0;
  float time_copy_out = 0;
  float time_kernel = 0;
  float time_malloc = 0;
  float time_free = 0;

  TIMESTAMP(t0);

  // Initialize data
  if (size) {
    if (args.unified) {
      checkCudaErrors(hipMallocManaged(&m, sizeof(float) * size * size));
    } else {
      m = (float*) malloc(sizeof(float) * size * size);
    }
    if (!m) {
      fprintf(stderr, "Failed to allocate memory: %s\n", strerror(errno));
      exit(EXIT_FAILURE);
    }
    TIMESTAMP(t1);
    time_malloc += ELAPSED(t0, t1);

    VPRINT(args.verbose, "Creating matrix internally size=%lu\n", size);
    const float lamda = -0.001;
    float coe[2 * size - 1];
    float coe_i = 0.0;

    for (int i = 0; i < size; i++) {
      coe_i = 10 * exp(lamda * i);
      int j = size - 1 + i;
      coe[j] = coe_i;
      j = size - 1 - i;
      coe[j] = coe_i;
    }

    for (int i = 0; i < size; i++) {
      for (int j = 0; j < size; j++) {
        m[i * size + j] = coe[size - 1 - i + j];
      }
    }
    TIMESTAMP(t2);
    time_pre += ELAPSED(t1, t2);
  } else {
    // File input
    VPRINT(args.verbose, "Reading matrix from file %s\n", args.file);

    FILE* fp = fopen(args.file, "rb");
    if (!fp) {
      fprintf(stderr, "Failed to open file: %s. %s\n", args.file, strerror(errno));
      exit(EXIT_FAILURE);
    }
    int ret = fscanf(fp, "%lu\n", &size);
    if (!ret) {
      fprintf(stderr, "Improperly formatted input file: %s\n", strerror(errno));
      exit(EXIT_FAILURE);
    }

    TIMESTAMP(t1);
    time_pre += ELAPSED(t0, t1);

    if (args.unified) {
      checkCudaErrors(hipMallocManaged(&m, sizeof(float) * size * size));
    } else {
      m = (float*) malloc(sizeof(float) * size * size);
    }
    if (!m) {
      fprintf(stderr, "Failed to allocate memory: %s\n", strerror(errno));
      exit(EXIT_FAILURE);
    }
    TIMESTAMP(t2);
    time_malloc += ELAPSED(t1, t2);

    for (int i = 0; i < size; i++) {
      for (int j = 0; j < size; j++) {
        ret = fscanf(fp, "%f ", &m[i * size + j]);
        if (!ret) {
          fprintf(stderr, "Improperly formatted input file. Input ended early. %s\n",
          strerror(errno));
          exit(EXIT_FAILURE);
        }
      }
    }
    fclose(fp);
    TIMESTAMP(t3);
    time_pre += ELAPSED(t2, t3);
  }

  TIMESTAMP(t1);
  if (!args.unified) {
    hipMalloc((void**) &d_m, size * size * sizeof(float));
    assert(d_m);
  }
  TIMESTAMP(t2);
  time_malloc += ELAPSED(t1, t2);

  if (args.unified) {
    d_m = m;
  } else {
    checkCudaErrors(hipMemcpyAsync(d_m, m, size * size * sizeof(float),
        hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipStreamSynchronize(stream));
  }
  TIMESTAMP(t3);
  time_copy_in += ELAPSED(t2, t3);

  time_kernel = lud_cuda(d_m, size, stream);

  TIMESTAMP(t4);
  if (!args.unified) {
    checkCudaErrors(hipMemcpyAsync(m, d_m, size * size * sizeof(float),
        hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));
  }
  TIMESTAMP(t5);
  time_copy_out += ELAPSED(t4, t5);

  // TODO something different
  // Access all data to bring it back to the host
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      m[i * size + j] += 1;
    }
  }
  TIMESTAMP(t6);
  time_post += ELAPSED(t5, t6);

  if (args.unified) {
    hipFree(m);
  } else {
    hipFree(d_m);
    free(m);
  }
  TIMESTAMP(t7);
  time_free += ELAPSED(t6, t7);

  printf("====Timing info====\n");
  printf("time malloc = %f ms\n", time_malloc);
  printf("time pre = %f ms\n", time_pre);
  printf("time copyIn = %f ms\n", time_copy_in);
  printf("time kernel = %f ms\n", time_kernel);
  printf("time serial = %f ms\n", time_serial);
  printf("time copyOut = %f ms\n", time_copy_out);
  printf("time post = %f ms\n", time_post);
  printf("time free = %f ms\n", time_free);
  printf("time end-to-end = %f ms\n", ELAPSED(t0, t7));
  exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                                                                //
//                                        Kernel Caller                                           //
//                                                                                                //
////////////////////////////////////////////////////////////////////////////////////////////////////
long long lud_cuda(float *m, int size, hipStream_t stream) {
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  float *m_debug = (float*) malloc(size * size * sizeof(float));

  int i;
  TIMESTAMP(start);
  for (i = 0; i < size - BLOCK_SIZE; i += BLOCK_SIZE) {
    lud_diagonal<<<1, BLOCK_SIZE, 0, stream>>>(m, size, i);
    checkCudaErrors(hipStreamSynchronize(stream));
    lud_perimeter<<<(size - i) / BLOCK_SIZE - 1, BLOCK_SIZE * 2, 0, stream>>>(m, size,
        i);
    checkCudaErrors(hipStreamSynchronize(stream));
    dim3 dimGrid((size - i) / BLOCK_SIZE - 1, (size - i) / BLOCK_SIZE - 1);
    lud_internal<<<dimGrid, dimBlock, 0, stream>>>(m, size, i);
    checkCudaErrors(hipStreamSynchronize(stream));
  }
  lud_diagonal<<<1, BLOCK_SIZE, 0, stream>>>(m, size, i);
  checkCudaErrors(hipStreamSynchronize(stream));
  TIMESTAMP(stop);
  return ELAPSED(start, stop);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                                                                //
//                                        Helper Functions                                        //
//                                                                                                //
////////////////////////////////////////////////////////////////////////////////////////////////////

void print_matrix(float *m, int size) {
  int i, j;
  for (i = 0; i < size; i++) {
    for (j = 0; j < size; j++)
      printf("%f ", m[i * size + j]);
    printf("\n");
  }
}
