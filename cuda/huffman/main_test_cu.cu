/*
 * PAVLE - Parallel Variable-Length Encoder for CUDA. Main file.
 *
 * Copyright (C) 2009 Ana Balevic <ana.balevic@gmail.com>
 * All rights reserved.
 *
 * This program is free software; you can redistribute it and/or modify it under the terms of the
 * MIT License. Read the full licence: http://www.opensource.org/licenses/mit-license.php
 *
 * If you find this program useful, please contact me and reference PAVLE home page in your work.
 *
 */

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "print_helpers.h"
#include "comparison_helpers.h"
#include "stats_logger.h"
#include "load_data.h"
#include <sys/time.h>
#include "vlc_kernel_sm64huff.cu"
#include "scan.cu"
#include "pack_kernels.cu"
#include "cpuencode.h"

long long get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec * 1000000) + tv.tv_usec;
}
void runVLCTest(char *file_name, uint num_block_threads, uint num_blocks=1, bool unified=false);

extern "C" void cpu_vlc_encode(unsigned int* indata, unsigned int num_elements, unsigned int* outdata, unsigned int *outsize, unsigned int *codewords, unsigned int* codewordlens);

int main(int argc, char* argv[]){
  unsigned int num_block_threads = 256;
  if (argc > 2) {
    bool unified = argv[1];
    for (int i=2; i<argc; i++) {
      runVLCTest(argv[i], num_block_threads, unified);
    }
  } else {
    bool unified = false;
    if (argc == 2) {
      unified = argv[1];
    }
    runVLCTest(NULL, num_block_threads, 1024, unified);
  }
  checkCudaErrors(hipDeviceReset());
  return 0;
}

void runVLCTest(char *file_name, uint num_block_threads, uint num_blocks, bool unified) {
  printf("CUDA! Starting VLC Tests!\n");
  unsigned int num_elements; //uint num_elements = num_blocks * num_block_threads;
  unsigned int mem_size; //uint mem_size = num_elements * sizeof(int);
  unsigned int symbol_type_size = sizeof(int);
  //////// LOAD DATA ///////////////
  double H; // entropy
  initParams(file_name, num_block_threads, num_blocks, num_elements, mem_size, symbol_type_size);
  printf("Parameters: num_elements: %d, num_blocks: %d, num_block_threads: %d\n----------------------------\n", num_elements, num_blocks, num_block_threads);
  ////////LOAD DATA ///////////////
  uint	*sourceData;
  uint	*destData;
  uint	*crefData;
  crefData=	(uint*) malloc(mem_size);
  uint	*codewords;
  uint	*codewordlens;
  if (unified) {
    checkCudaErrors(hipMallocManaged(&sourceData, mem_size));
    checkCudaErrors(hipMallocManaged(&destData, mem_size));
    checkCudaErrors(hipMallocManaged(&codewords, NUM_SYMBOLS * symbol_type_size));
    checkCudaErrors(hipMallocManaged(&codewordlens, NUM_SYMBOLS * symbol_type_size));
  } else {
    sourceData =	(uint*) malloc(mem_size);
    destData =	(uint*) malloc(mem_size);
    codewords = (uint*) malloc(NUM_SYMBOLS*symbol_type_size);
    codewordlens = (uint*) malloc(NUM_SYMBOLS*symbol_type_size);
  }

  uint	*cw32 =		(uint*) malloc(mem_size);
  uint	*cw32len =	(uint*) malloc(mem_size);
  uint	*cw32idx =	(uint*) malloc(mem_size);

  uint	*cindex2=	(uint*) malloc(num_blocks*sizeof(int));

  memset(sourceData,   0, mem_size);
  memset(destData,     0, mem_size);
  memset(crefData,     0, mem_size);
  memset(cw32,         0, mem_size);
  memset(cw32len,      0, mem_size);
  memset(cw32idx,      0, mem_size);
  memset(codewords,    0, NUM_SYMBOLS*symbol_type_size);
  memset(codewordlens, 0, NUM_SYMBOLS*symbol_type_size);
  memset(cindex2, 0, num_blocks*sizeof(int));
  //////// LOAD DATA ///////////////
  loadData(file_name, sourceData, codewords, codewordlens, num_elements, mem_size, H);

  //////// LOAD DATA ///////////////

  unsigned int	*d_sourceData, *d_destData, *d_destDataPacked;
  unsigned int	*d_codewords, *d_codewordlens;
  unsigned int	*d_cw32, *d_cw32len, *d_cw32idx, *d_cindex, *d_cindex2;

  if (unified) {
    checkCudaErrors(hipMallocManaged((void**) &d_destDataPacked,	  mem_size));
  } else {
    checkCudaErrors(hipMalloc((void**) &d_sourceData,		  mem_size));
    checkCudaErrors(hipMalloc((void**) &d_destData,			  mem_size));
    checkCudaErrors(hipMalloc((void**) &d_destDataPacked,	  mem_size));

    checkCudaErrors(hipMalloc((void**) &d_codewords,		  NUM_SYMBOLS*symbol_type_size));
    checkCudaErrors(hipMalloc((void**) &d_codewordlens,		  NUM_SYMBOLS*symbol_type_size));
  }

  checkCudaErrors(hipMalloc((void**) &d_cw32,				  mem_size));
  checkCudaErrors(hipMalloc((void**) &d_cw32len,			  mem_size));
  checkCudaErrors(hipMalloc((void**) &d_cw32idx,			  mem_size));

  checkCudaErrors(hipMalloc((void**)&d_cindex,         num_blocks*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void**)&d_cindex2,        num_blocks*sizeof(unsigned int)));

  if (unified) {
    d_sourceData = sourceData;
    d_codewords = codewords;
    d_codewordlens = codewordlens;
    d_destData = destData;
  } else {
    checkCudaErrors(hipMemcpy(d_sourceData,		sourceData,		mem_size,		hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_codewords,		codewords,		NUM_SYMBOLS*symbol_type_size,	hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_codewordlens,	codewordlens,	NUM_SYMBOLS*symbol_type_size,	hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_destData,		destData,		mem_size,		hipMemcpyHostToDevice));
  }

  dim3 grid_size(num_blocks,1,1);
  dim3 block_size(num_block_threads, 1, 1);
  unsigned int sm_size;


  unsigned int NT = 10; //number of runs for each execution time

  //////////////////* CPU ENCODER *///////////////////////////////////
  unsigned int refbytesize;
  long long timer = get_time();
  cpu_vlc_encode((unsigned int*)sourceData, num_elements, (unsigned int*)crefData,  &refbytesize, codewords, codewordlens);
  float msec = (float)((get_time() - timer)/1000.0);
  printf("CPU Encoding time (CPU): %f (ms)\n", msec);
  printf("CPU Encoded to %d [B]\n", refbytesize);
  unsigned int num_ints = refbytesize/4 + ((refbytesize%4 ==0)?0:1);
  //////////////////* END CPU *///////////////////////////////////

  //////////////////* SM64HUFF KERNEL *///////////////////////////////////
  grid_size.x		= num_blocks;
  block_size.x	= num_block_threads;
  sm_size			= block_size.x*sizeof(unsigned int);
#ifdef CACHECWLUT
  sm_size			= 2*NUM_SYMBOLS*sizeof(int) + block_size.x*sizeof(unsigned int);
#endif
  hipEvent_t     start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord( start, 0 );
  for (int i=0; i<NT; i++) {
    vlc_encode_kernel_sm64huff<<<grid_size, block_size, sm_size>>>(d_sourceData, d_codewords, d_codewordlens,
#ifdef TESTING
        d_cw32, d_cw32len, d_cw32idx,
#endif
        d_destData, d_cindex); //testedOK2
  }
  hipDeviceSynchronize();
  hipEventRecord( stop, 0 ) ;
  hipEventSynchronize( stop ) ;
  float   elapsedTime;
  hipEventElapsedTime( &elapsedTime,
      start, stop ) ;

  printf("GPU Encoding time (SM64HUFF): %f (ms)\n", elapsedTime/NT);
  //////////////////* END KERNEL *///////////////////////////////////

#ifdef TESTING
  unsigned int num_scan_elements = grid_size.x;
  preallocBlockSums(num_scan_elements);
  hipMemset(d_destDataPacked, 0, mem_size);
  printf("Num_blocks to be passed to scan is %d.\n", num_scan_elements);
  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
  prescanArray(d_cindex2, d_cindex, num_scan_elements, stream);

  pack2<<< num_scan_elements/16, 16, 0, stream>>>((unsigned int*)d_destData, d_cindex, d_cindex2, (unsigned int*)d_destDataPacked, num_elements/num_scan_elements);
  checkCudaErrors(hipStreamSynchronize(stream));
  deallocBlockSums();

  checkCudaErrors(hipMemcpy(destData, d_destDataPacked, mem_size, hipMemcpyDeviceToHost));
  compare_vectors((unsigned int*)crefData, (unsigned int*)destData, num_ints);
#endif

  if (unified) {
    checkCudaErrors(hipFree(sourceData));
    checkCudaErrors(hipFree(destData));
    checkCudaErrors(hipFree(codewords));
    checkCudaErrors(hipFree(codewordlens));
  } else {
    free(sourceData); free(destData);  	free(codewords);  	free(codewordlens); free(cw32);  free(cw32len); free(crefData);
    checkCudaErrors(hipFree(d_sourceData)); 	checkCudaErrors(hipFree(d_destData)); checkCudaErrors(hipFree(d_destDataPacked));
    checkCudaErrors(hipFree(d_codewords)); 		checkCudaErrors(hipFree(d_codewordlens));
  }
  checkCudaErrors(hipFree(d_cw32)); 		checkCudaErrors(hipFree(d_cw32len)); 	checkCudaErrors(hipFree(d_cw32idx));
  checkCudaErrors(hipFree(d_cindex)); checkCudaErrors(hipFree(d_cindex2));
  free(cindex2);
}

