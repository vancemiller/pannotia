/*
 * PAVLE - Parallel Variable-Length Encoder for CUDA. Main file.
 *
 * Copyright (C) 2009 Ana Balevic <ana.balevic@gmail.com>
 * All rights reserved.
 *
 * This program is free software; you can redistribute it and/or modify it under the terms of the
 * MIT License. Read the full licence: http://www.opensource.org/licenses/mit-license.php
 *
 * If you find this program useful, please contact me and reference PAVLE home page in your work.
 *
 */

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "print_helpers.h"
#include "comparison_helpers.h"
#include "stats_logger.h"
#include "load_data.h"
#include <sys/time.h>
#include "vlc_kernel_sm64huff.cu"
#include "scan.cu"
#include "pack_kernels.cu"
#include "cpuencode.h"

#include "../timing.h"

void runVLCTest(char *file_name, uint num_block_threads, bool unified=false, uint num_blocks=1);

extern "C" void cpu_vlc_encode(unsigned int* indata, unsigned int num_elements, unsigned int* outdata, unsigned int *outsize, unsigned int *codewords, unsigned int* codewordlens);

int main(int argc, char* argv[]){
  unsigned int num_block_threads = 256;
  if (argc > 2) {
    bool unified = (bool) (atoi(argv[1]) != 0);
    for (int i=2; i<argc; i++) {
      runVLCTest(argv[i], num_block_threads, unified);
    }
  } else if (argc == 2) {
    bool unified = (bool) (atoi(argv[1]) != 0);
    runVLCTest(NULL, num_block_threads, 1024, unified);
  }
  checkCudaErrors(hipDeviceReset());
  return 0;
}

void runVLCTest(char *file_name, uint num_block_threads, bool unified, uint num_blocks) {
  float time_pre = 0;
  float time_post = 0;
  float time_serial = 0;
  float time_copy_in = 0;
  float time_copy_out = 0;
  float time_kernel = 0;
  float time_malloc = 0;
  float time_free = 0;
  printf("CUDA! Starting VLC Tests!\n");
  unsigned int num_elements; //uint num_elements = num_blocks * num_block_threads;
  unsigned int mem_size; //uint mem_size = num_elements * sizeof(int);
  unsigned int symbol_type_size = sizeof(int);
  //////// LOAD DATA ///////////////
  double H; // entropy
  TIMESTAMP(t0);
  initParams(file_name, num_block_threads, num_blocks, num_elements, mem_size, symbol_type_size);
  printf("Parameters: num_elements: %d, num_blocks: %d, num_block_threads: %d\n----------------------------\n", num_elements, num_blocks, num_block_threads);
  TIMESTAMP(t1);
  time_pre += ELAPSED(t0, t1);
  ////////LOAD DATA ///////////////
  uint	*sourceData;
  uint	*destData;
  uint	*crefData;
  crefData=	(uint*) malloc(mem_size);
  uint	*codewords;
  uint	*codewordlens;
  if (unified) {
    checkCudaErrors(hipMallocManaged(&sourceData, mem_size));
    checkCudaErrors(hipMallocManaged(&destData, mem_size));
    checkCudaErrors(hipMallocManaged(&codewords, NUM_SYMBOLS * symbol_type_size));
    checkCudaErrors(hipMallocManaged(&codewordlens, NUM_SYMBOLS * symbol_type_size));
  } else {
    sourceData =	(uint*) malloc(mem_size);
    destData =	(uint*) malloc(mem_size);
    codewords = (uint*) malloc(NUM_SYMBOLS*symbol_type_size);
    codewordlens = (uint*) malloc(NUM_SYMBOLS*symbol_type_size);
  }

  uint	*cw32 =		(uint*) malloc(mem_size);
  uint	*cw32len =	(uint*) malloc(mem_size);
  uint	*cw32idx =	(uint*) malloc(mem_size);

  uint	*cindex2=	(uint*) malloc(num_blocks*sizeof(int));

  TIMESTAMP(t2);
  time_malloc += ELAPSED(t1, t2);

  memset(sourceData,   0, mem_size);
  memset(destData,     0, mem_size);
  memset(crefData,     0, mem_size);
  memset(cw32,         0, mem_size);
  memset(cw32len,      0, mem_size);
  memset(cw32idx,      0, mem_size);
  memset(codewords,    0, NUM_SYMBOLS*symbol_type_size);
  memset(codewordlens, 0, NUM_SYMBOLS*symbol_type_size);
  memset(cindex2, 0, num_blocks*sizeof(int));
  TIMESTAMP(t3);
  time_pre += ELAPSED(t2, t3);

  //////// LOAD DATA ///////////////
  loadData(file_name, sourceData, codewords, codewordlens, num_elements, mem_size, H);

  //////// LOAD DATA ///////////////
  TIMESTAMP(t3p);

  unsigned int	*d_sourceData, *d_destData, *d_destDataPacked;
  unsigned int	*d_codewords, *d_codewordlens;
  unsigned int	*d_cw32, *d_cw32len, *d_cw32idx, *d_cindex, *d_cindex2;

  if (unified) {
    checkCudaErrors(hipMallocManaged((void**) &d_destDataPacked,	  mem_size));
  } else {
    checkCudaErrors(hipMalloc((void**) &d_sourceData,		  mem_size));
    checkCudaErrors(hipMalloc((void**) &d_destData,			  mem_size));
    checkCudaErrors(hipMalloc((void**) &d_destDataPacked,	  mem_size));

    checkCudaErrors(hipMalloc((void**) &d_codewords,		  NUM_SYMBOLS*symbol_type_size));
    checkCudaErrors(hipMalloc((void**) &d_codewordlens,		  NUM_SYMBOLS*symbol_type_size));
  }

  checkCudaErrors(hipMalloc((void**) &d_cw32,				  mem_size));
  checkCudaErrors(hipMalloc((void**) &d_cw32len,			  mem_size));
  checkCudaErrors(hipMalloc((void**) &d_cw32idx,			  mem_size));

  checkCudaErrors(hipMalloc((void**)&d_cindex,         num_blocks*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void**)&d_cindex2,        num_blocks*sizeof(unsigned int)));

  TIMESTAMP(t4);
  time_malloc += ELAPSED(t3, t4);

  if (unified) {
    d_sourceData = sourceData;
    d_codewords = codewords;
    d_codewordlens = codewordlens;
    d_destData = destData;
  } else {
    checkCudaErrors(hipMemcpy(d_sourceData,		sourceData,		mem_size,		hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_codewords,		codewords,		NUM_SYMBOLS*symbol_type_size,	hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_codewordlens,	codewordlens,	NUM_SYMBOLS*symbol_type_size,	hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_destData,		destData,		mem_size,		hipMemcpyHostToDevice));
  }
  TIMESTAMP(t5);
  time_copy_in += ELAPSED(t4, t5);

  dim3 grid_size(num_blocks,1,1);
  dim3 block_size(num_block_threads, 1, 1);
  unsigned int sm_size;


  unsigned int NT = 10; //number of runs for each execution time

  //////////////////* CPU ENCODER *///////////////////////////////////
  unsigned int refbytesize;
  cpu_vlc_encode((unsigned int*)sourceData, num_elements, (unsigned int*)crefData,  &refbytesize, codewords, codewordlens);
  unsigned int num_ints = refbytesize/4 + ((refbytesize%4 ==0)?0:1);
  //////////////////* END CPU *///////////////////////////////////

  //////////////////* SM64HUFF KERNEL *///////////////////////////////////
  grid_size.x		= num_blocks;
  block_size.x	= num_block_threads;
  sm_size			= block_size.x*sizeof(unsigned int);
#ifdef CACHECWLUT
  sm_size			= 2*NUM_SYMBOLS*sizeof(int) + block_size.x*sizeof(unsigned int);
#endif
  hipEvent_t     start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  TIMESTAMP(t6);
  time_serial += ELAPSED(t5, t6);

  hipEventRecord( start, 0 );
  for (int i=0; i<NT; i++) {
    vlc_encode_kernel_sm64huff<<<grid_size, block_size, sm_size>>>(d_sourceData, d_codewords, d_codewordlens,
        d_cw32, d_cw32len, d_cw32idx,
        d_destData, d_cindex); //testedOK2
  }
  hipDeviceSynchronize();
  hipEventRecord( stop, 0 ) ;
  hipEventSynchronize( stop ) ;
  TIMESTAMP(t7);
  time_kernel += ELAPSED(t6, t7);
  float elapsedTime;
  hipEventElapsedTime( &elapsedTime,
      start, stop ) ;

  printf("CUDA-reported GPU Encoding time (SM64HUFF): %f (ms)\n", elapsedTime/NT);
  //////////////////* END KERNEL *///////////////////////////////////

  unsigned int num_scan_elements = grid_size.x;
  preallocBlockSums(num_scan_elements);
  hipMemset(d_destDataPacked, 0, mem_size);
  printf("Num_blocks to be passed to scan is %d.\n", num_scan_elements);
  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
  prescanArray(d_cindex2, d_cindex, num_scan_elements, stream);

  pack2<<< num_scan_elements/16, 16, 0, stream>>>((unsigned int*)d_destData, d_cindex, d_cindex2, (unsigned int*)d_destDataPacked, num_elements/num_scan_elements);
  checkCudaErrors(hipStreamSynchronize(stream));
  TIMESTAMP(t8);
  time_kernel += ELAPSED(t7, t8);
  deallocBlockSums();
  TIMESTAMP(t9);
  time_free += ELAPSED(t8, t9);

  checkCudaErrors(hipMemcpy(destData, d_destDataPacked, mem_size, hipMemcpyDeviceToHost));
  TIMESTAMP(t10);
  time_copy_out += ELAPSED(t9, t10);

  compare_vectors((unsigned int*)crefData, (unsigned int*)destData, num_ints);
  TIMESTAMP(t11);
  time_post += ELAPSED(t10, t11);

  if (unified) {
    checkCudaErrors(hipFree(sourceData));
    checkCudaErrors(hipFree(destData));
    checkCudaErrors(hipFree(codewords));
    checkCudaErrors(hipFree(codewordlens));
  } else {
    free(sourceData); free(destData);  	free(codewords);  	free(codewordlens); free(cw32);  free(cw32len); free(crefData);
    checkCudaErrors(hipFree(d_sourceData)); 	checkCudaErrors(hipFree(d_destData)); checkCudaErrors(hipFree(d_destDataPacked));
    checkCudaErrors(hipFree(d_codewords)); 		checkCudaErrors(hipFree(d_codewordlens));
  }
  checkCudaErrors(hipFree(d_cw32)); 		checkCudaErrors(hipFree(d_cw32len)); 	checkCudaErrors(hipFree(d_cw32idx));
  checkCudaErrors(hipFree(d_cindex)); checkCudaErrors(hipFree(d_cindex2));
  free(cindex2);
  TIMESTAMP(t12);
  time_free += ELAPSED(t11, t12);

  printf("====Timing info====\n");
  printf("time malloc = %f ms\n", time_malloc);
  printf("time pre = %f ms\n", time_pre);
  printf("time copyIn = %f ms\n", time_copy_in);
  printf("time kernel = %f ms\n", time_kernel);
  printf("time serial = %f ms\n", time_serial);
  printf("time copyOut = %f ms\n", time_copy_out);
  printf("time post = %f ms\n", time_post);
  printf("time free = %f ms\n", time_free);
  printf("time end-to-end = %f ms\n", ELAPSED(t0, t12));
  exit(EXIT_SUCCESS);
}

