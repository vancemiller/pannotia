#include "hip/hip_runtime.h"
#include <errno.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>
#include <stdlib.h>
#include <math.h>

#define BLOCK_SIZE 16
#define STR_SIZE 256

#define block_x_ 128
#define block_y_ 2
#define block_z_ 1
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5


#define TIMESTAMP(NAME) \
  struct timespec NAME; \
if (clock_gettime(CLOCK_MONOTONIC, &NAME)) { \
  fprintf(stderr, "Failed to get time: %s\n", strerror(errno)); \
}

#define ELAPSED(start, end) \
  ((long long int) 1e9 * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec)

/*timing globals */
long long time_pre = 0;
long long time_post = 0;
long long time_serial = 0;
long long time_copy_in = 0;
long long time_copy_out = 0;
long long time_kernel = 0;
long long time_malloc = 0;
long long time_free = 0;

#include "opt1.cu"

/* chip parameters	*/
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016; /* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0;

void fatal(const char *s) {
  fprintf(stderr, "Error: %s\n", s);
}

void readinput(float *vect, int grid_rows, int grid_cols, int layers, char *file) {
  int i, j, k;
  FILE *fp;
  char str[STR_SIZE];
  float val;

  if ((fp = fopen(file, "r")) == 0)
    fatal("The file was not opened");

  for (i = 0; i <= grid_rows - 1; i++)
    for (j = 0; j <= grid_cols - 1; j++)
      for (k = 0; k <= layers - 1; k++) {
        if (fgets(str, STR_SIZE, fp) == NULL)
          fatal("Error reading file\n");
        if (feof(fp))
          fatal("not enough lines in file");
        if ((sscanf(str, "%f", &val) != 1))
          fatal("invalid file format");
        vect[i * grid_cols + j + k * grid_rows * grid_cols] = val;
      }

  fclose(fp);

}

void writeoutput(float *vect, int grid_rows, int grid_cols, int layers, char *file) {
  int i, j, k, index = 0;
  FILE *fp;
  char str[STR_SIZE];

  if ((fp = fopen(file, "w")) == 0)
    printf("The file was not opened\n");

  for (i = 0; i < grid_rows; i++)
    for (j = 0; j < grid_cols; j++)
      for (k = 0; k < layers; k++) {
        sprintf(str, "%d\t%g\n", index, vect[i * grid_cols + j + k * grid_rows * grid_cols]);
        fputs(str, fp);
        index++;
      }

  fclose(fp);
}

void computeTempCPU(float *pIn, float* tIn, float *tOut, int nx, int ny, int nz, float Cap,
    float Rx, float Ry, float Rz, float dt, int numiter) {
  float ce, cw, cn, cs, ct, cb, cc;
  float stepDivCap = dt / Cap;
  ce = cw = stepDivCap / Rx;
  cn = cs = stepDivCap / Ry;
  ct = cb = stepDivCap / Rz;

  cc = 1.0 - (2.0 * ce + 2.0 * cn + 3.0 * ct);

  int c, w, e, n, s, b, t;
  int x, y, z;
  int i = 0;
  do {
    for (z = 0; z < nz; z++)
      for (y = 0; y < ny; y++)
        for (x = 0; x < nx; x++) {
          c = x + y * nx + z * nx * ny;

          w = (x == 0) ? c : c - 1;
          e = (x == nx - 1) ? c : c + 1;
          n = (y == 0) ? c : c - nx;
          s = (y == ny - 1) ? c : c + nx;
          b = (z == 0) ? c : c - nx * ny;
          t = (z == nz - 1) ? c : c + nx * ny;

          tOut[c] = tIn[c] * cc + tIn[n] * cn + tIn[s] * cs + tIn[e] * ce + tIn[w] * cw
            + tIn[t] * ct + tIn[b] * cb + (dt / Cap) * pIn[c] + ct * amb_temp;
        }
    float *temp = tIn;
    tIn = tOut;
    tOut = temp;
    i++;
  } while (i < numiter);
}

float accuracy(float *arr1, float *arr2, int len) {
  float err = 0.0;
  int i;
  for (i = 0; i < len; i++) {
    err += (arr1[i] - arr2[i]) * (arr1[i] - arr2[i]);
  }

  return (float) sqrt(err / len);
}

void usage(int argc, char **argv) {
  fprintf(stderr,
      "Usage: %s <rows/cols> <layers> <iterations> <powerFile> <tempFile> <outputFile>\n", argv[0]);
  fprintf(stderr, "\t<rows/cols>  - number of rows/cols in the grid (positive integer)\n");
  fprintf(stderr, "\t<layers>  - number of layers in the grid (positive integer)\n");

  fprintf(stderr, "\t<iteration> - number of iterations\n");
  fprintf(stderr,
      "\t<powerFile>  - name of the file containing the initial power values of each cell\n");
  fprintf(stderr,
      "\t<tempFile>  - name of the file containing the initial temperature values of each cell\n");
  fprintf(stderr, "\t<outputFile - output file\n");
  fprintf(stderr, "\t<optional unified flag> - unified memory\n");
  exit(1);
}

int main(int argc, char** argv) {
  if (argc < 7) {
    usage(argc, argv);
  }

  char *pfile, *tfile, *ofile;
  int iterations = atoi(argv[3]);

  pfile = argv[4];
  tfile = argv[5];
  ofile = argv[6];
  bool unified = argc == 8;
  int numCols = atoi(argv[1]);
  int numRows = atoi(argv[1]);
  int layers = atoi(argv[2]);

  /* calculating parameters*/

  TIMESTAMP(t0);
  float dx = chip_height / numRows;
  float dy = chip_width / numCols;
  float dz = t_chip / layers;

  float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * dx * dy;
  float Rx = dy / (2.0 * K_SI * t_chip * dx);
  float Ry = dx / (2.0 * K_SI * t_chip * dy);
  float Rz = dz / (K_SI * dx * dy);

  float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
  float dt = PRECISION / max_slope;

  float *powerIn, *tempOut, *tempIn, *tempCopy;
  int size = numCols * numRows * layers;

  TIMESTAMP(t1);
  time_pre += ELAPSED(t0, t1);
  if (unified) {
    hipMallocManaged(&powerIn, size * sizeof(float));
    hipMallocManaged(&tempIn, size * sizeof(float));
    hipMallocManaged(&tempOut, size * sizeof(float));
  } else {
    powerIn = (float*) calloc(size, sizeof(float));
    tempIn = (float*) calloc(size, sizeof(float));
    tempOut = (float*) calloc(size, sizeof(float));
  }
  tempCopy = (float*) malloc(size * sizeof(float));
  float* answer = (float*) calloc(size, sizeof(float));
  TIMESTAMP(t2);
  time_malloc += ELAPSED(t1, t2);

  TIMESTAMP(t3);
  readinput(powerIn, numRows, numCols, layers, pfile);
  readinput(tempIn, numRows, numCols, layers, tfile);

  memcpy(tempCopy, tempIn, size * sizeof(float));
  TIMESTAMP(t4);
  time_pre += ELAPSED(t3, t4);

  hotspot_opt1(powerIn, tempIn, tempOut, numCols, numRows, layers, Cap, Rx, Ry, Rz, dt, iterations,
      unified);

  TIMESTAMP(t5);
  computeTempCPU(powerIn, tempCopy, answer, numCols, numRows, layers, Cap, Rx, Ry, Rz, dt,
      iterations);
  TIMESTAMP(t6);
  time_serial += ELAPSED(t5, t6);

  float acc = accuracy(tempOut, answer, numRows * numCols * layers);
  printf("Accuracy: %e\n", acc);
  writeoutput(tempOut, numRows, numCols, layers, ofile);
  TIMESTAMP(t7);
  time_post += ELAPSED(t6, t7);

  if (unified) {
    hipFree(tempIn);
    hipFree(tempOut);
    hipFree(powerIn);
  } else {
    free(tempIn);
    free(tempOut);
    free(powerIn);
  }
  TIMESTAMP(t8);
  time_free += ELAPSED(t7, t8);

  printf("====Timing info====\n");
  printf("time malloc = %f ms\n", time_malloc * 1e-6);
  printf("time pre = %f ms\n", time_pre * 1e-6);
  printf("time CPU to GPU memory copy = %f ms\n", time_copy_in * 1e-6);
  printf("time kernel = %f ms\n", time_kernel * 1e-6);
  printf("time serial = %f ms\n", time_serial * 1e-6);
  printf("time GPU to CPU memory copy back = %f ms\n", time_copy_out * 1e-6);
  printf("time post = %f ms\n", time_post * 1e-6);
  printf("time free = %f ms\n", time_free * 1e-6);
  printf("End-to-end = %f ms\n", ELAPSED(t0, t8) * 1e-6);
  exit(EXIT_SUCCESS);
}

